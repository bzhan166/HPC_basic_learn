#include<iostream>
#include<hip/hip_runtime.h>
#include<hipblas.h>

#include <sys/time.h>

void vector_init(float *a, int n){
    for(int i = 0; i < n; i++){
        a[i] = float(rand() % 1000)/1000;
    }
}

int main(){
    int n = 1 <<20;//1048576
    int bytes = n *sizeof(float);
    float *h_a, *h_b;
    float *d_a, *d_b;
    float result = 0.0;

    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);

    hipMallocManaged(&d_a, bytes);
    hipMallocManaged(&d_b, bytes);

    vector_init(h_a, n);
    vector_init(h_b, n);

    // cudaEvent used to record the time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //use for cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //copy the vectors over to the device
    hipblasSetVector(n, sizeof(float), h_a, 1, d_a, 1);
    hipblasSetVector(n, sizeof(float), h_b, 1, d_b, 1);


    //do cublas sdot
    hipEventRecord(start);
    hipblasSdot(handle,n,d_a,1,d_b,1,&result);
    hipEventRecord(stop);

    //get the running time milliseconds
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float diff = 0.0f;
    float sum = 0.0f;

    for(int i = 0; i < n; i++){
        sum += h_a[i]*h_b[i]; 
    }
    diff = fabs(sum - result);
        
    
    std::cout<<"h_a[1]= "<<h_a[0]<<" "<<"h_b[1]= "<<h_b[0]<<" "<<"result= "<<result<<" "<<std::endl;
    std::cout << "Diff: " << diff << std::endl;
    std::cout << "Time:" << milliseconds << std::endl;
    std::cout<< "Performace: "<< (2*n) /milliseconds/1000000.0<< std::endl;

    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);

    hipblasDestroy(handle);

    return 0;
}